#include "hip/hip_runtime.h"
#include "kernels.h"
#include "assert.h"

// https://github.com/AlexeyAB/darknet/blob/master/src/blas_kernels.cu
__global__ void scale_channels_kernel(float *in_w_h_c, int size, int channel_size, int batch_size, int scale_wh, float *scales_c, float *out)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    if (index < size) {
        if (scale_wh) {
            int osd_index = index % channel_size + (index / batch_size)*channel_size;

            out[index] = in_w_h_c[index] * scales_c[osd_index];
        }
        else {
            out[index] = in_w_h_c[index] * scales_c[index / channel_size];
        }
    }
}

void scaleChannelsForward(dnnType *in_w_h_c, int size, int channel_size, int batch_size, int scale_wh, 
    dnnType *scales_c, dnnType *out, hipStream_t stream)
{
    int blocks = (size+255)/256;
    int threads = 256;

    scale_channels_kernel <<<blocks, threads, 0, stream>>>(in_w_h_c, size, channel_size, batch_size, scale_wh, scales_c, out);
}
